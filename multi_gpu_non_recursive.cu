//#include "scale.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <math.h> /* fabs */
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include "omp.h"
#include <unordered_map>

using namespace std;
#define THREADS_PER_BLOCK 256
#define PARALEL_THREAD_COUNT 4

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


void printArray(int *arr, int nov){
    for (int j=0;j<nov;j++)
    {
        cout << j << " " << arr[j] << endl;
    }
}
// __device__ bool check(int marked[], int round, int val){
//   for(int i = 0; i < round; i++){
//     if(marked[i] == val){return false;}
//   }
//   return true;
// }
//
// __device__ void DFS_sparse(int xadj[], int adj[], int marked[], int n,
//          int vert, int start, int &count, int round) //vert: bulundugu konum //start: baslangıc noktası
// {
//     marked[round] = vert;
//
//     int start_index = xadj[vert];
//     int path_length = xadj[vert+1];
//
//     if (n == 0){
//       marked[round] = -1;
//         for(int i = start_index; i < path_length; i++){
//             if(adj[i] == start){
//                 count++;
//                 break;
//             }
//         }
//         return;
//     }
//
//     for(int i=start_index; i < path_length; i++){
//         if(check(marked, round,adj[i])){
//             DFS_sparse(xadj, adj, marked, n-1, adj[i], start, count, round + 1);
//         }
//     }
//     marked[round] = -1;
// }


__global__ void kernel3(int* adj, int* xadj, int* output, int nov, int novStart){
  int index = novStart + threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3
            int neighbour_3 =  adj[k];
            if (neighbour_3 == index){
              localcount+=1;
              break;
            }
          }
        }
      }
      output[index-novStart] = localcount;
    }
}

__global__ void kernel4(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              if (neighbour_4 == index){
                localcount+=1;
                break;
              }
            }
          }
        }
      }
      output[index] = localcount;
    }
}
__global__ void kernel5(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              //eliminate 4 == 0
              if (neighbour_4 == index) continue;
              // eliminate 4 ==1
              if (neighbour_4 == neighbour_1) continue;
              // eliminate 4 ==2
              if (neighbour_4 == neighbour_2) continue;

              int s4  = xadj[neighbour_4];
              int e4  = xadj[neighbour_4+1];


              for(int o=s4; o < e4; o++){

                //0 -->1 -->2 -->3 -->4--> 5

                int neighbour_5 =  adj[o];
                if (neighbour_5 == index){
                  localcount+=1;
                  break;
                }
              }
            }
          }
        }
      }
      output[index] = localcount;
    }
}


void wrapper(int *xadj, int *adj, int n,  int nov, int nnz){

  int *output_h = new int[nov];



  #pragma omp parallel num_threads(PARALEL_THREAD_COUNT)
  {

    int threadId=omp_get_thread_num ();
    // cout<< threadId<<endl;
    int novForThread = (nov+PARALEL_THREAD_COUNT-1)/PARALEL_THREAD_COUNT;
    int novStart = novForThread * threadId;
    int novEnd   = novForThread * (threadId+1);
    if (novEnd> nov) novEnd = nov;
    int numBlock = (novEnd-novStart + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    if(threadId)



    hipSetDevice(threadId);
    int *adj_d;
    int *xadj_d;
    int *output_d;
    hipEvent_t start, stop;
    float elapsedTime;

    gpuErrchk(hipMalloc((void**)&adj_d, (nnz) * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&xadj_d, (nov + 1) * sizeof(int)));

    gpuErrchk(hipMalloc((void**)&output_d, (novEnd-novStart) * sizeof(int)));

    //gpuErrchk(cudaMallocHost((void **)&output_h, (nov) * sizeof(int)));

    gpuErrchk(hipMemcpy(adj_d, adj, (nnz) * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(xadj_d, xadj, (nov + 1) * sizeof(int), hipMemcpyHostToDevice));

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    if      (n==3)kernel3<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, novEnd,novStart);
    else if (n==4)kernel4<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, novEnd);
    else if (n==5)kernel5<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, novEnd);
    //combination<<<numBlocks, threadsPerBlock>>>(adj_d, xadj_d, output_d, n, nov);

    gpuErrchk(hipDeviceSynchronize());
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("GPU scale took: %f s on gpu %d\n", elapsedTime/1000, threadId);

    gpuErrchk(hipMemcpy(output_h+novStart, output_d, (novEnd-novStart) * sizeof(int), hipMemcpyDeviceToHost));

    hipFree(adj_d);
    hipFree(xadj_d);

  }
  // printArray(output_h,nov);
}


void  read_mtxbin(string fname, int k){
  //cout << "fname: " << fname << endl;
  ifstream infile(fname);
  int a, b;
  int nnv = 0;
  unordered_map<int, vector<int> > hashmap;

  int  maxElement  = -1;

  while (infile >> a >> b)
  {
      nnv+=2;
      hashmap[a].push_back(b);
      hashmap[b].push_back(a);

      if(b > maxElement){
        maxElement = b;
      }
  }
  //cout << end1-start1 << " -- ILK OKUMA SU (s).\n";

  int nov = maxElement +1;
  //cout <<"nov " << nov << endl;
  //cout <<"nnv " << nnv << endl;
  int * adj  = new int[nnv];
  int * xadj = new int[nov+1];
  xadj[0]=0;

  int j = 0;
  int maxSize = -1;

  for(int i=0; i < nov ; i++ ){
    auto current = hashmap.find(i);
    if (current == hashmap.end()){
        xadj[i+1] = xadj[i];
    }
    else{
        int size = current->second.size();
        maxSize = max(size,maxSize);

        xadj[i+1] = xadj[i] + size;
        for(auto val : current->second) {
            adj[j] = val;
            j++;
        }
    }
  }
  // cout << "maxSize: "<<maxSize<<endl;
  // cout << end-start << " -- OKUMA SURE (s).\n";
  wrapper(xadj,adj,k,nov,nnv);
  //cout<<"CYCLES: --> "<<countCycles_sparse(xadj, adj,k,nov)<<endl;

  /*double end2 = omp_get_wtime();
  cout << end2-start << " -- TOTAL SURE (s).\n";*/
}

int main(int argc, char *argv[]){
    char* fname = argv[1];
    int k = atoi(argv[2]);
    read_mtxbin(fname,k);
    return 0;
}
